#include "hip/hip_runtime.h"

#include <hip/hip_runtime_api.h>

#include "hair_kernel.cu"

static float* mallocFloat(const int bytes)
{
	float* pointer;
	
	checkCudaErrors(hipMalloc((void**)&pointer, bytes));
	checkCudaErrors(hipMemset(pointer, 0, bytes));
	
	return pointer;
}

static float3* mallocFloat3(const int bytes)
{
	float3* pointer;
	
	checkCudaErrors(hipMalloc((void**)&pointer, bytes));
	checkCudaErrors(hipMemset(pointer, 0, bytes));
	
	return pointer;
}

extern "C"
void mallocStrands(const int &numStrands,
				   const int &numParticles,
				   const int &numComponents,
				   float3* &root,
				   float3* &normal,
				   float3* &position,
				   float3* &pos,
				   float3* &posc,
				   float3* &posh,
				   float3* &velocity,
				   float3* &velh,
				   float3* &force,
				   float* &AA,
				   float* &bb,
				   float* &xx)
{
	int bytes1D = numParticles * numStrands * numComponents * sizeof(float);
	int bytes2D = numParticles * numStrands * numComponents * numParticles * numStrands * numComponents * sizeof(float);
	int bytes3fR = numStrands * sizeof(float3);
	int bytes3f1D = numParticles * numStrands * sizeof(float3);
	
	root	 = mallocFloat3(bytes3fR);
	normal	 = mallocFloat3(bytes3fR);
	//~ position = mallocFloat3(bytes3f1D);
	pos		 = mallocFloat3(bytes3f1D);
	posc	 = mallocFloat3(bytes3f1D);
	posh	 = mallocFloat3(bytes3f1D);
	velocity = mallocFloat3(bytes3f1D);
	velh	 = mallocFloat3(bytes3f1D);
	force	 = mallocFloat3(bytes3f1D);
	
	AA = mallocFloat(bytes2D);
	
	bb = mallocFloat(bytes1D);
	xx = mallocFloat(bytes1D);
}

extern "C"
void freeStrands(float3* &root,
				 float3* &normal,
				 float3* &position,
				 float3* &pos,
				 float3* &posc,
				 float3* &posh,
				 float3* &velocity,
				 float3* &velh,
				 float3* &force,
				 float* &AA,
				 float* &bb,
				 float* &xx)
{
	checkCudaErrors(hipFree(root));
	checkCudaErrors(hipFree(normal));
	//~ checkCudaErrors(hipFree(position));
	checkCudaErrors(hipFree(pos));
	checkCudaErrors(hipFree(posc));
	checkCudaErrors(hipFree(posh));
	checkCudaErrors(hipFree(velocity));
	checkCudaErrors(hipFree(velh));
	checkCudaErrors(hipFree(force));
	checkCudaErrors(hipFree(AA));
	checkCudaErrors(hipFree(bb));
	checkCudaErrors(hipFree(xx));
}

extern "C"
void copyRoots(int numStrands, const float3* root3f, const float3* normal3f, float3* root, float3* normal)
{
	int size = numStrands * sizeof(float3);
	checkCudaErrors(hipMemcpy(root, root3f, size, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(normal, normal3f, size, hipMemcpyHostToDevice));
}

extern "C"
void initPositions(int numStrands, int numParticles, const float3* root, float3* normal, float3* position, float3* posc, float3* pos)
{
	dim3 grid(numStrands, 1, 1);
	dim3 block(1, 1, 1);
	
	initialise<<<grid,block>>>(numParticles, root, normal, position, posc, pos);
	
	hipDeviceSynchronize();
}

extern "C"
void updateStrandsNew(int numParticles,
					  int numStrands,
					  int numComponents,
					  float dt,
					  float mass,
					  float k_edge,
					  float k_bend,
					  float k_twist,
					  float k_extra,
					  float d_edge,
					  float d_bend,
					  float d_twist,
					  float d_extra,
					  float length_e,
					  float length_b,
					  float length_t,
					  float3 &gravity,
					  float3* root,
					  float3* position,
					  float3* posc,
					  float3* posh,
					  float3* pos,
					  float3* velocity,
					  float3* velh,
					  float3* force,
					  float* AA,
					  float* bb,
					  float* xx)
{
	dim3 grid(numStrands, 1, 1);
	dim3 block(1, 1, 1);
//	static bool once = false;
	
//	if(!once)
//	{
		update_strands<<<grid,block>>>(numParticles,
				numStrands,
				numComponents,
				dt,
				mass,
				k_edge,
				k_bend,
				k_twist,
				k_extra,
				d_edge,
				d_bend,
				d_twist,
				d_extra,
				length_e,
				length_b,
				length_t,
				gravity,
				root,
				position,
				posc,
				posh,
				pos,
				velocity,
				velh,
				force,
				AA,
				bb,
				xx);

		hipDeviceSynchronize();

//		once = true;
//	}
}


#include <hip/hip_runtime_api.h>

#include "hair.h"
#include "hair_kernel.cu"

static
void* mallocBytes(int bytes)
{
	void* pointer;
	
	//Allocate bytes of memory
	checkCudaErrors(hipMalloc((void**)&pointer, bytes));
	
	//Set memory to zero
	checkCudaErrors(hipMemset(pointer, 0, bytes));
	
	return pointer;
}

extern "C"
void mallocStrands(pilar::HairState* h_state, pilar::HairState* &d_state, int modelBytes)
{
	h_state->AA = (float*) mallocBytes(h_state->numParticles * h_state->numStrands * h_state->numComponents * h_state->numParticles * h_state->numStrands * h_state->numComponents * sizeof(float));
	h_state->bb = (float*) mallocBytes(h_state->numParticles * h_state->numStrands * h_state->numComponents * sizeof(float));
	h_state->xx = (float*) mallocBytes(h_state->numParticles * h_state->numStrands * h_state->numComponents * sizeof(float));

	h_state->root	  = (pilar::Vector3f*) mallocBytes(h_state->numStrands * sizeof(pilar::Vector3f));
	h_state->normal	  = (pilar::Vector3f*) mallocBytes(h_state->numStrands * sizeof(pilar::Vector3f));
	//~ h_state->position = (pilar::Vector3f*) mallocBytes(numParticles * numStrands * sizeof(pilar::Vector3f));
	h_state->pos	  = (pilar::Vector3f*) mallocBytes(h_state->numParticles * h_state->numStrands * sizeof(pilar::Vector3f));
	h_state->posc	  = (pilar::Vector3f*) mallocBytes(h_state->numParticles * h_state->numStrands * sizeof(pilar::Vector3f));
	h_state->posh	  = (pilar::Vector3f*) mallocBytes(h_state->numParticles * h_state->numStrands * sizeof(pilar::Vector3f));
	h_state->velocity = (pilar::Vector3f*) mallocBytes(h_state->numParticles * h_state->numStrands * sizeof(pilar::Vector3f));
	h_state->velh 	  = (pilar::Vector3f*) mallocBytes(h_state->numParticles * h_state->numStrands * sizeof(pilar::Vector3f));
	h_state->force	  = (pilar::Vector3f*) mallocBytes(h_state->numParticles * h_state->numStrands * sizeof(pilar::Vector3f));
	
	h_state->rng	  = (hiprandStatePhilox4_32_10_t*) mallocBytes(h_state->numStrands * sizeof(hiprandStatePhilox4_32_10_t));
	
	h_state->vertices = (float*) mallocBytes(modelBytes);
	h_state->normals  = (float*) mallocBytes(modelBytes*sizeof(float));
	h_state->faces	  = (float*) mallocBytes(modelBytes*sizeof(float));
	h_state->model	  = (ModelOBJ*) mallocBytes(sizeof(ModelOBJ));
	
	d_state = (pilar::HairState*) mallocBytes(sizeof(pilar::HairState));
}

extern "C"
void freeStrands(pilar::HairState* h_state, pilar::HairState* d_state)
{
	checkCudaErrors(hipFree(h_state->AA));
	checkCudaErrors(hipFree(h_state->bb));
	checkCudaErrors(hipFree(h_state->xx));

	checkCudaErrors(hipFree(h_state->root));
	checkCudaErrors(hipFree(h_state->normal));
	//~ checkCudaErrors(hipFree(h_state->position));
	checkCudaErrors(hipFree(h_state->pos));
	checkCudaErrors(hipFree(h_state->posc));
	checkCudaErrors(hipFree(h_state->posh));
	checkCudaErrors(hipFree(h_state->velocity));
	checkCudaErrors(hipFree(h_state->velh));
	checkCudaErrors(hipFree(h_state->force));
	
	checkCudaErrors(hipFree(h_state->rng));
	
	checkCudaErrors(hipFree(h_state->vertices));
	checkCudaErrors(hipFree(h_state->normals));
	checkCudaErrors(hipFree(h_state->faces));
	checkCudaErrors(hipFree(h_state->model));
	
	checkCudaErrors(hipFree(d_state));
}

extern "C"
void copyRoots(pilar::Vector3f* roots, pilar::Vector3f* normals, pilar::HairState* h_state)
{
	checkCudaErrors(hipMemcpy(h_state->root,   roots,   h_state->numStrands * sizeof(*roots), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(h_state->normal, normals, h_state->numStrands * sizeof(*normals), hipMemcpyHostToDevice));
}

extern "C"
void copyState(pilar::HairState* h_state, pilar::HairState* d_state)
{
	checkCudaErrors(hipMemcpy(d_state, h_state, sizeof(*h_state), hipMemcpyHostToDevice));
}

extern "C"
void copyModel(ModelOBJ* model, pilar::HairState* h_state)
{
	checkCudaErrors(hipMemcpy(h_state->model,	  model,		   sizeof(*model), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(h_state->vertices, model->vertices, model->bytes,			   hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(h_state->normals,  model->normals,  model->bytes*sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(h_state->faces,	  model->faces,    model->bytes*sizeof(float), hipMemcpyHostToDevice));
}

extern "C"
void initialisePositions(pilar::HairState* h_state, pilar::HairState* d_state)
{
	dim3 grid(h_state->numStrands, 1, 1);
	dim3 block(1, 1, 1);
	
	initialise<<<grid,block>>>(d_state);
	
	hipDeviceSynchronize();
}

extern "C"
void updateStrands(float dt, pilar::HairState* h_state, pilar::HairState* d_state)
{
	dim3 grid(h_state->numStrands, 1, 1);
	dim3 block(1, 1, 1);
//	static bool once = false;
//
//	if(!once)
//	{
		update<<<grid,block>>>(dt, d_state);
		
		hipDeviceSynchronize();

//		once = true;
//	}
}


#include <hip/hip_runtime_api.h>

#include "hair.h"
#include "hair_kernel.cu"

static void* mallocBytes(int bytes)
{
	void* pointer;
	
	//Allocate bytes of memory
	checkCudaErrors(hipMalloc((void**)&pointer, bytes));
	
	//Set memory to zero
	checkCudaErrors(hipMemset(pointer, 0, bytes));

	return pointer;
}

extern "C"
void mallocStrands(pilar::HairState* h_state, pilar::HairState* &d_state)
{
	h_state->AA = (float*) mallocBytes(h_state->numParticles * h_state->numStrands * h_state->numComponents * h_state->numParticles * h_state->numStrands * h_state->numComponents * sizeof(float));
	h_state->bb = (float*) mallocBytes(h_state->numParticles * h_state->numStrands * h_state->numComponents * sizeof(float));
	h_state->xx = (float*) mallocBytes(h_state->numParticles * h_state->numStrands * h_state->numComponents * sizeof(float));

	h_state->root	  = (pilar::Vector3f*) mallocBytes(h_state->numStrands * sizeof(pilar::Vector3f));
	h_state->normal	  = (pilar::Vector3f*) mallocBytes(h_state->numStrands * sizeof(pilar::Vector3f));
	//~ h_state->position = (pilar::Vector3f*) mallocBytes(numParticles * numStrands * sizeof(pilar::Vector3f));
	h_state->pos	  = (pilar::Vector3f*) mallocBytes(h_state->numParticles * h_state->numStrands * sizeof(pilar::Vector3f));
	h_state->posc	  = (pilar::Vector3f*) mallocBytes(h_state->numParticles * h_state->numStrands * sizeof(pilar::Vector3f));
	h_state->posh	  = (pilar::Vector3f*) mallocBytes(h_state->numParticles * h_state->numStrands * sizeof(pilar::Vector3f));
	h_state->velocity = (pilar::Vector3f*) mallocBytes(h_state->numParticles * h_state->numStrands * sizeof(pilar::Vector3f));
	h_state->velh 	  = (pilar::Vector3f*) mallocBytes(h_state->numParticles * h_state->numStrands * sizeof(pilar::Vector3f));
	h_state->force	  = (pilar::Vector3f*) mallocBytes(h_state->numParticles * h_state->numStrands * sizeof(pilar::Vector3f));

	d_state = (pilar::HairState*) mallocBytes(sizeof(pilar::HairState));
}

extern "C"
void freeStrands(pilar::HairState* h_state, pilar::HairState* d_state)
{
	checkCudaErrors(hipFree(h_state->AA));
	checkCudaErrors(hipFree(h_state->bb));
	checkCudaErrors(hipFree(h_state->xx));

	checkCudaErrors(hipFree(h_state->root));
	checkCudaErrors(hipFree(h_state->normal));
	//~ checkCudaErrors(hipFree(h_state->position));
	checkCudaErrors(hipFree(h_state->pos));
	checkCudaErrors(hipFree(h_state->posc));
	checkCudaErrors(hipFree(h_state->posh));
	checkCudaErrors(hipFree(h_state->velocity));
	checkCudaErrors(hipFree(h_state->velh));
	checkCudaErrors(hipFree(h_state->force));

	checkCudaErrors(hipFree(d_state));
}

extern "C"
void copyRoots(pilar::Vector3f* roots, pilar::Vector3f* normals, pilar::HairState* h_state)
{
	checkCudaErrors(hipMemcpy(h_state->root,   roots,   h_state->numStrands * sizeof(pilar::Vector3f), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(h_state->normal, normals, h_state->numStrands * sizeof(pilar::Vector3f), hipMemcpyHostToDevice));
}

extern "C"
void copyState(pilar::HairState* h_state, pilar::HairState* d_state)
{
	checkCudaErrors(hipMemcpy(d_state, h_state, sizeof(pilar::HairState), hipMemcpyHostToDevice));
}

extern "C"
void initialisePositions(pilar::HairState* h_state, pilar::HairState* d_state)
{
	dim3 grid(h_state->numStrands, 1, 1);
	dim3 block(1, 1, 1);
	
	initialise<<<grid,block>>>(d_state);
	
	hipDeviceSynchronize();
}

extern "C"
void updateStrands(float dt, pilar::HairState* h_state, pilar::HairState* d_state)
{
	dim3 grid(h_state->numStrands, 1, 1);
	dim3 block(1, 1, 1);
//	static bool once = false;
//
//	if(!once)
//	{
		update<<<grid,block>>>(dt, d_state);
		
		hipDeviceSynchronize();

//		once = true;
//	}
}

#include "hip/hip_runtime.h"

//#include <cutil_inline.h>

#include <hip/hip_runtime_api.h>

#include "hair_kernel.cu"

static float3* init(const int size)
{
	float3* d_vec;
	checkCudaErrors(hipMalloc((void**)&d_vec, size));
	checkCudaErrors(hipMemset(d_vec, 0, size));
	
	return d_vec;
}

static float* init2(const int size)
{
	float* d_vec;
	checkCudaErrors(hipMalloc((void**)&d_vec, size));
	checkCudaErrors(hipMemset(d_vec, 0, size));
	
	return d_vec;
}

extern "C"
void initStrands(int numStrands,
				 int numParticles,
				 float length,
				 const float3 *root,
				 float3* &position,
				 float3* &posc,
				 float3* &posh,
				 float3* &velocity,
				 float3* &velc,
				 float3* &velh,
				 float3* &force,
				 float* &A,
				 float* &b,
				 float* &x,
				 float* &r,
				 float* &p,
				 float* &Ap)
{
	int size = numStrands*numParticles*sizeof(float3);
	
//	position = init(size);
	posc = init(size);
	posh = init(size);
	velocity = init(size);
	velh = init(size);
	velc = init(size);
	force = init(size);
	A = init2(numStrands*numParticles*3*numParticles*3*sizeof(float));
	b = init2(numStrands*numParticles*3*sizeof(float));
	x = init2(numStrands*numParticles*3*sizeof(float));
	r = init2(numStrands*numParticles*3*sizeof(float));
	p = init2(numStrands*numParticles*3*sizeof(float));
	Ap = init2(numStrands*numParticles*3*sizeof(float));
	
	//TODO posh posc pos poso position
	//TODO velh velc velocity
	//TODO force
	//TODO AA bb xx
	
	
	/*
	float3* position_h = (float3*) calloc(numStrands*numParticles, sizeof(float3));
	
	for(int i = 0; i < numStrands; i++)
	{
		for(int j = 0; j < numParticles; j++)
		{
			int index = i*numParticles + j;
			position_h[index].x = root[i].x + j * length / 2.0f;
			position_h[index].y = root[i].y;
			position_h[index].z = root[i].z;
			
//			printf("%f %f %f\n", position_h[index].x, position_h[index].y, position_h[index].z);
		}
	}
	
	printf("before memcpy\n");
	checkCudaErrors(hipMemcpy(position, position_h, size, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(posc, position, size, hipMemcpyDeviceToDevice));
	printf("after memcpy\n");
	
	free(position_h);
	*/
}

extern "C"
void copyMem(const int numStrands,
			 const int numParticles,
			 float3* &position,
			 float3* &posc)
{
	checkCudaErrors(hipMemcpy(posc, position, numStrands*numParticles*sizeof(float3), hipMemcpyDeviceToDevice));
}


extern "C"
void releaseStrands(float3* &position,
				 	float3* &posc,
				 	float3* &posh,
				 	float3* &velocity,
				 	float3* &velc,
				 	float3* &velh,
				 	float3* &force,
				 	float* &A,
				 	float* &b,
				 	float* &x,
				 	float* &r,
				 	float* &p,
				 	float* &Ap)
{
	/*
	float3* position_h = (float3*) calloc(numStrands*numParticles, sizeof(float3));
	
	checkCudaErrors(hipMemcpy(position_h, position, numStrands*numParticles*sizeof(float3), hipMemcpyDeviceToHost));
	
	for(int i = 0; i < numStrands; i++)
	{
		for(int j = 0; j < numParticles; j++)
		{
			int index = i * numParticles + j;
			
			printf("%f %f %f\n", position_h[index].x, position_h[index].y, position_h[index].z);
		}
	}
	
	free(position_h);
	*/
	
	checkCudaErrors(hipFree(posc));
	checkCudaErrors(hipFree(posh));
	checkCudaErrors(hipFree(velocity));
	checkCudaErrors(hipFree(velc));
	checkCudaErrors(hipFree(velh));
	checkCudaErrors(hipFree(force));
	checkCudaErrors(hipFree(A));
	checkCudaErrors(hipFree(b));
	checkCudaErrors(hipFree(x));
	checkCudaErrors(hipFree(r));
	checkCudaErrors(hipFree(p));
	checkCudaErrors(hipFree(Ap));
//	checkCudaErrors(hipFree(position));
}

extern "C"
void updateStrands(const int numParticles,
				   float4 &mlgt,
				   const float4 k,
				   const float4 d,
				   float3* &position,
				   float3* &posc,
				   float3* &posh,
				   float3* &velocity,
				   float3* &velc,
				   float3* &velh,
				   float3* &force,
				   float* &A,
				   float* &b,
				   float* &x,
				   float* &r,
				   float* &p,
				   float* &Ap)
{
	dim3 grid(1,1,1);
	dim3 block(1,1,1);
	
	update<<<grid, block>>>(numParticles,
							mlgt,
							k,
							d,
							position,
							posc,
							posh,
							velocity,
							velc,
							velh,
							force,
							A,
							b,
							x,
							r,
							p,
							Ap);
	
	hipDeviceSynchronize();
}



